#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <tuple>
#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics.h"
#include "./headers/logger.h"

int block_size = 512;
int n_body;
int n_iteration;
double totalTime;

void generate_data(double *m, double *x,double *y,double *vx,double *vy, int n) {
    // Generate proper initial position and mass for better visualization
    srand((unsigned)time(NULL));
    for (int i = 0; i < n; i++) {
        m[i] = rand() % max_mass + 1.0f;
        x[i] = 2000.0f + rand() % (bound_x / 4);
        y[i] = 2000.0f + rand() % (bound_y / 4);
        vx[i] = 0.0f;
        vy[i] = 0.0f;
    }

}

__global__ void update_position(double *x, double *y, double *vx, double *vy, int n) {
    // Update position 
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        x[i] = x[i] + (vx[i] * dt);
        y[i] = y[i] + (vy[i] * dt);

        if (x[i] < 0) {
            vx[i] *= -1;
        } else if (x[i] >= bound_x){
            vx[i] *= -1; 
        } else if (y[i] < 0) {
            vy[i] *= -1;
        } else if (y[i] >= bound_y){
            vy[i] *= -1;
        }
    }
}

__global__ void update_velocity(double *m, double *x, double *y, double *vx, double *vy, int n) {
    // Calculate force and acel, update velocity
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {  
        for (size_t j = 0; j < n; j++) {
            double dx, dy, distance, ax, ay, acel;
            if (i == j) continue;
            dx = x[j] - x[i];
            dy = y[j] - y[i];

            distance = (dx * dx) + (dy * dy) + err;
            distance = sqrt(distance);

            if (distance <= radius2){
                distance = radius2;
            } else{
                distance = distance;
            }

            acel = (gravity_const * m[j]) / (distance * distance);
            ax = dx/distance * acel;
            ay = dy/distance * acel;
            if (distance == radius2 || distance <= radius2) {
                ax = 0;
                ay = 0;
            }
            vx[i] = vx[i] + (dt * ax);
            vy[i] = vy[i] + (dt * ay);
        }
    }
}



void master() {
    double* m = new double[n_body];
    double* x = new double[n_body];
    double* y = new double[n_body];
    double* vx = new double[n_body];
    double* vy = new double[n_body];

    generate_data(m, x, y, vx, vy, n_body);
    Logger l = Logger("CUDA", n_body, bound_x, bound_y);

    double *device_m;
    double *device_x;
    double *device_y;
    double *device_vx;
    double *device_vy;

    hipMalloc(&device_m, n_body * sizeof(double));
    hipMalloc(&device_x, n_body * sizeof(double));
    hipMalloc(&device_y, n_body * sizeof(double));
    hipMalloc(&device_vx, n_body * sizeof(double));
    hipMalloc(&device_vy, n_body * sizeof(double));

    hipMemcpy(device_m, m, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_x, x, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_y, y, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vx, vx, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vy, vy, n_body * sizeof(double), hipMemcpyHostToDevice);

    int n_block = n_body / block_size + 1;

    for (int i = 0; i < n_iteration; i++){
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

        update_velocity<<<n_block, block_size>>>(device_m, device_x, device_y, device_vx, device_vy, n_body);
        update_position<<<n_block, block_size>>>(device_x, device_y, device_vx, device_vy, n_body);

        hipMemcpy(x, device_x, n_body * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(y, device_y, n_body * sizeof(double), hipMemcpyDeviceToHost);

        std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> time_span = t2 - t1;
        
        printf("Iteration %d, elapsed time: %.3f\n", i, time_span);
        totalTime += time_span.count();

        l.save_frame(x, y);

        #ifdef GUI
        glClear(GL_COLOR_BUFFER_BIT);
        glColor3f(1.0f, 0.0f, 0.0f);
        glPointSize(2.0f);
        glBegin(GL_POINTS);
        double xi;
        double yi;
        for (int i = 0; i < n_body; i++){
            xi = x[i];
            yi = y[i];
            glVertex2f(xi, yi);
        }
        glEnd();
        glFlush();
        glutSwapBuffers();
        #else

        #endif

    }

    hipFree(device_m);
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_vx);
    hipFree(device_vy);

    delete[] m;
    delete[] x;
    delete[] y;
    delete[] vx;
    delete[] vy;
    
}


int main(int argc, char *argv[]){
    
    n_body = atoi(argv[1]);
    n_iteration = atoi(argv[2]);

    #ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(500, 500);
    glutCreateWindow("N Body Simulation CUDA Implementation");
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    gluOrtho2D(0, bound_x, 0, bound_y);
    #endif

    totalTime = 0.0;
    master();

    printf("Student ID: 119010507\n"); // replace it with your student id
    printf("Name: Ryan Christopher\n"); // replace it with your name
    printf("Assignment 2: N Body Simulation CUDA Implementation\n");
    printf("Total running time: %.4f\n", totalTime);

    return 0;

}